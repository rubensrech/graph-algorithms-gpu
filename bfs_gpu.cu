#include <iostream>
#include <fstream>
#include <stack>
#include <queue>
#include <sys/time.h>
using namespace std;

#include <hip/hip_runtime.h>

#define INF 99999
#define GOAL 5000

double GetTime(void)
{
   struct  timeval time;
   double  Time;
   
   gettimeofday(&time, (struct timezone *) NULL);
   Time = ((double)time.tv_sec*1000000.0 + (double)time.tv_usec);
   return(Time);
}

__global__ void bfs_kernel(int nNodes, int *graph, bool *visited, bool *running) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int v = 0; v < nNodes; v += num_threads) {
        int vertex = v + tid;
        if (vertex < nNodes) {
            for (int i = 0; i < nNodes; i++) {
                if (graph[vertex * nNodes + i] != INF && vertex != i) {
                    // Neighbor
                    if (!visited[i]) {
                        visited[i] = true;
                        *running = true;
                    }
                }
            }
        }
    }
}


int main(int argc, char **argv){
	double timeElapsed, clockBegin;
	int **graph;
    int a, b, w, nNodes;

    if (argc < 2) {
        cout << "Usage: ./" << argv[0] << " <graph>" << endl;
        exit(-1);
    }

    /* Initialization */
    ifstream inputfile(argv[1]);
    inputfile >> nNodes;
    graph = new int*[nNodes];
    for (int i = 0; i < nNodes; ++i)
    {
        graph[i] = new int[nNodes]; 
        for (int j = 0; j < nNodes; ++j)
            graph[i][j] = INF;
    }
    while (inputfile >> a >> b >> w)
    {
        graph[a][b] = w;
        graph[b][a] = w;
    }

    /* BFS */
    int *graph_d;
    bool false_val = false;
    bool *running, *running_d;
    bool *visited, *visited_d;

    hipMalloc(&graph_d, nNodes * nNodes * sizeof(int));
    for (int i = 0; i < nNodes; i++)
        hipMemcpy(&graph_d[i * nNodes], graph[i], nNodes * sizeof(int), hipMemcpyHostToDevice);

    running = new bool[1];
    *running = true;
    hipMalloc(&running_d, 1 * sizeof(bool));

    visited = new bool[nNodes];
    for(int i = 0; i < nNodes; i++)
        visited[i] = 0;
    hipMalloc(&visited_d, nNodes * sizeof(bool));
    hipMemcpy(visited_d, visited, nNodes * sizeof(bool), hipMemcpyHostToDevice);

    clockBegin = GetTime();
    
	while (*running) {
        hipMemcpy(running_d, &false_val, 1 * sizeof(bool), hipMemcpyHostToDevice);
        int blockSize = 32;
        int numBlocks = (nNodes + blockSize - 1) / blockSize;
        bfs_kernel<<<numBlocks, blockSize>>>(nNodes, graph_d, visited_d, running_d);
        hipMemcpy(running, running_d, 1 * sizeof(bool), hipMemcpyDeviceToHost);
    }

    timeElapsed = (GetTime() - clockBegin)/1000000;
    
    hipMemcpy(visited, visited_d, nNodes * sizeof(bool), hipMemcpyDeviceToHost);
    for (int i = 0; i < nNodes; i++)
        cout << "node " << i << ": " << visited[i] << endl;
        
    printf("Total time: %5lf\n", timeElapsed);

	for(int i = 0; i < nNodes; i++)
		free(graph[i]);
	free(visited);
}

